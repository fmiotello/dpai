#include "hip/hip_runtime.h"
#include <torch/types.h>
#include <ATen/cuda/HIPContext.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <string>

#define CUDA_MAX_THREADS 1024 // this is safe.

namespace{
    template <typename scalar_t>
    __global__ void interp_affine_out_kernel(
        const torch::PackedTensorAccessor<scalar_t,4,torch::RestrictPtrTraits,size_t> idata,
        torch::PackedTensorAccessor<scalar_t,4,torch::RestrictPtrTraits,size_t> odata,
        const torch::PackedTensorAccessor<int,1,torch::RestrictPtrTraits,size_t> index,
        const torch::PackedTensorAccessor<scalar_t,1,torch::RestrictPtrTraits,size_t> weight,
        const int num_kernels,
        const int k,
        const int n
    ){
        const int batchsize = idata.size(0);
        const int channelsize = idata.size(1);
        const int freqsize = idata.size(2);
        const int timesize = idata.size(3);

        const int id = blockIdx.x * blockDim.x + threadIdx.x;
        const int time_id = id % timesize;
        const int freq_id = id / timesize;

        const int step = freq_id / n;
        const int nn = freq_id % n;

        const int ref_index_0 = step * k + index[nn];
        const int ref_index_1 = ref_index_0 + 1;
        const float weight_0 = weight[nn];
        const float weight_1 = 1 - weight_0;

        if (id < num_kernels && ref_index_1 < freqsize){
            for (int b = 0; b < batchsize; b++){
                for (int c = 0; c < channelsize; c++){
                    odata[b][c][freq_id][time_id] = weight_0 * idata[b][c][ref_index_0][time_id] + weight_1 * idata[b][c][ref_index_1][time_id]; 
                }
            }
        }
    }

    template <typename scalar_t>
    __global__ void interp_shift_out_kernel(
        const torch::PackedTensorAccessor<scalar_t,4,torch::RestrictPtrTraits,size_t> idata,
        torch::PackedTensorAccessor<scalar_t,4,torch::RestrictPtrTraits,size_t> odata,
        const int top_to_target,
        const float bottom_weight,
        const int num_kernels
    ){
        const int batchsize = idata.size(0);
        const int channelsize = idata.size(1);
        const int freqsize = idata.size(2);
        const int timesize = idata.size(3);

        const int id = blockIdx.x * blockDim.x + threadIdx.x;
        const int time_id = id % timesize;
        const int freq_id = id / timesize;

        const int tf = freq_id - top_to_target;
        const int bf = freq_id - top_to_target - 1;   
        const float top_weight = 1 - bottom_weight;//0.2
        // shift = 1.8
        // 3    1.2  top
        // 2    0.2
        // 1 -> 0
        // 0    0    bottom

        if (id < num_kernels && bf >= 0 && tf < freqsize){ 
            for (int b = 0; b < batchsize; b++){
                for (int c = 0; c < channelsize; c++){                
                    odata[b][c][freq_id][time_id] = bottom_weight * idata[b][c][bf][time_id] + top_weight * idata[b][c][tf][time_id];
                }
            }
        }
    }

}// namespace

void interp_affine_out_cuda(
    torch::Tensor input,
    torch::Tensor output,
    torch::Tensor indexes,
    torch::Tensor weights,
    int k,
    int n
){
    // input & output [batch,channel,freq,time]
    const int num_kernels = input.size(2)*input.size(3);
    
    // const int threads = std::min<int>(
        // at::cuda::getCurrentDeviceProperties()->maxThreadsPerBlock, CUDA_MAX_THREADS);
    // I don't know why, but in my environment the above line malfunctions.
    const int threads = 1024;
    const dim3 blocks((num_kernels - 1)/threads + 1);
    hipStream_t stream = at::cuda::getCurrentCUDAStream();

    AT_DISPATCH_FLOATING_TYPES(
        input.scalar_type(), "interp_affine_cuda", [&] {
    
            auto idata = input.packed_accessor<scalar_t, 4, torch::RestrictPtrTraits,size_t>();
            auto odata = output.packed_accessor<scalar_t, 4, torch::RestrictPtrTraits,size_t>();
            auto index_data = indexes.packed_accessor<int, 1, torch::RestrictPtrTraits,size_t>();
            auto weight_data = weights.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits,size_t>();
    
            interp_affine_out_kernel<scalar_t>
                <<<blocks, threads, 0, stream>>>(idata, odata, index_data, weight_data, num_kernels, k, n);
        }
    );
    AT_CUDA_CHECK(hipGetLastError());
}

void interp_shift_out_cuda(
    torch::Tensor input,
    torch::Tensor output,
    float shift
){
    // input & output [batch,channel,freq,time]
    const int num_kernels = input.size(2) * input.size(3);
    const int threads = 1024;
    const dim3 blocks((num_kernels - 1) / threads + 1);

    hipStream_t stream = at::cuda::getCurrentCUDAStream();

    const int top_to_target = (int) std::floor(shift);//1
    const float bottom_weight = shift - (float) top_to_target;//0.8
    // shift = 1.8
    // 3    1.2
    // 2    0.2
    // 1 -> 0
    // 0    0
    AT_DISPATCH_FLOATING_TYPES(
        input.scalar_type(), "interp_shift_cuda", [&] {
    
            auto idata = input.packed_accessor<scalar_t, 4, torch::RestrictPtrTraits,size_t>();
            auto odata = output.packed_accessor<scalar_t, 4, torch::RestrictPtrTraits,size_t>();
    
            interp_shift_out_kernel<scalar_t>
                <<<blocks, threads, 0, stream>>>(idata, odata, top_to_target, bottom_weight, num_kernels);
        }
    );
    AT_CUDA_CHECK(hipGetLastError());
}
